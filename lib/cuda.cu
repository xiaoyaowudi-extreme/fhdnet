#include <cuda.hpp>

namespace fhdnet
{
    hipDeviceProp_t cuda_device_property;
    bool is_cuda_initted = false;
    void cuda_init()
    {
        if(is_cuda_initted) return;
        CHECK( hipGetDeviceProperties(&cuda_device_property, 0) );
        CHECK( hipSetDevice(0) );
        is_cuda_initted = true;
    }
}